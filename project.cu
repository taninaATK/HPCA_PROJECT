
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

//A TRIER
#define EPS 0.0000001f
#define r 0.1f
#define N 3			// Size of the problem's matrix
#define NB 2		// Number of blocks
#define NTPB 3		// Number of threads per block
#define MIN 1
#define MAX 10

void printVect(float *v, int n){
	for(int i = 0; i < n; i++){
		printf("%f ", v[i]);
	}
	printf("\n\n");
}

// Used 
float randFloat(){
	return ((MAX - MIN) * ((float)rand() / RAND_MAX)) + MIN;
}

// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line)  {
	if (error != hipSuccess) {
	printf("There is an error in file %s at line %d\n", file, line);
	printf("%s\n", hipGetErrorName(error));
	printf("%s\n", hipGetErrorString(error));
	exit(EXIT_FAILURE);
	} 
}
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

void iniTest(float* a,float* b, float* c, float* y, int n){
	// Initialisation d'un système de taille n pour les tests
	for(int i = 0; i < n; i++){
		// POur éviter l'overflow
		a[i] = randFloat();
		c[i] = randFloat();
		b[i] = randFloat();
		y[i] = randFloat();
	}
}

// CREDITS TO Lokman A. Abbas-Turki FOR THIS CODE
__device__ void PCR_d(float* sa, float* sd, float* sc, float* sy, int* sl, int n) {
	int i, lL, d, tL, tR;
	float aL, dL, cL, yL;
	float aLp, dLp, cLp, yLp;

	d = (n / 2 + (n % 2)) * (threadIdx.x % 2) + (int)threadIdx.x / 2;

	tL = threadIdx.x - 1;
	if (tL < 0) tL = 0;
	tR = threadIdx.x + 1;
	if (tR >= n) tR = 0;

	for (i = 0; i < (int)(logf((float)n) / logf(2.0f)) + 1; i++) {
		lL = (int)sl[threadIdx.x];

		aL = sa[threadIdx.x];
		dL = sd[threadIdx.x];
		cL = sc[threadIdx.x];
		yL = sy[threadIdx.x];

		dLp = sd[tL];
		cLp = sc[tL];

		if (fabsf(aL) > EPS) {
			aLp = sa[tL];
			yLp = sy[tL];
			dL -= aL * cL / dLp;
			yL -= aL * yLp / dLp;
			aL = -aL * aLp / dLp;
			cL = -cLp * cL / dLp;
		}

		cLp = sc[tR];
		if (fabsf(cLp) > EPS) {
			aLp = sa[tR];
			dLp = sd[tR];
			yLp = sy[tR];
			dL -= cLp * aLp / dLp;
			yL -= cLp * yLp / dLp;
		}
		__syncthreads();

		if (i < (int)(logf((float)n) / logf(2.0f))) {
			sa[d] = aL;
			sd[d] = dL;
			sc[d] = cL;
			sy[d] = yL;
			sl[d] = (int)lL;
			__syncthreads();
		}
	}

	sy[(int)sl[threadIdx.x]] = yL / dL;
}

__global__ void PCR(float* sa, float* sd, float* sc, float* sy, int* sl, int n){
	for (int i = 0; i <= n; i++) {
		sl[threadIdx.x] = threadIdx.x;
	}
	__syncthreads();
	PCR_d(sa, sd, sc, sy, sl, n);
}

/*
*/
__global__ void Thomas(float* aGPU, float* bGPU, float* cGPU, float* yGPU, float* zGPU, int n){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	//Forward step
	cGPU[(n-1)*idx] = cGPU[(n-1)*idx]/bGPU[n*idx];
	yGPU[n*idx] = yGPU[n*idx]/bGPU[n*idx];

	for(int i = 1; i < n; i++){
		if(i < n-1) cGPU[i + (n-1)*idx] = cGPU[i + (n-1)*idx]/(bGPU[i + n*idx] - aGPU[i - 1 + (n-1)*idx]*cGPU[i-1 + (n-1)*idx]);
		yGPU[i + n*idx] = (yGPU[i + n*idx] - aGPU[i - 1 + (n-1)*idx]*yGPU[i - 1 + n*idx])/(bGPU[i + n*idx] - aGPU[i - 1+ (n-1)*idx]*cGPU[i - 1 + (n-1)*idx]);
	}

	//Backward step
	zGPU[n-1 + n*idx] = yGPU[n-1 + n*idx];
	for(int i = n-2; i >= 0; i--){
		zGPU[i + n*idx] = yGPU[i + n*idx] - cGPU[i + (n-1)*idx] * zGPU[i+1 + n*idx];
	}
}

void Thomas_wrap(float* a, float* b, float* c, float* y, float* z, int n){
	// Déclaration des variables utilisées
	float *aGPU, *bGPU, *cGPU, *yGPU, *zGPU;

	// Allocation des vecteurs dans la mémoire GPU
	testCUDA(hipMalloc(&aGPU, NB*NTPB*(n-1)*sizeof(float)));
	testCUDA(hipMalloc(&cGPU, NB*NTPB*(n-1)*sizeof(float)));
	testCUDA(hipMalloc(&bGPU, NB*NTPB*n*sizeof(float)));
	testCUDA(hipMalloc(&yGPU, NB*NTPB*n*sizeof(float)));
	testCUDA(hipMalloc(&zGPU, NB*NTPB*n*sizeof(float)));

	// Copie des données dans les vecteurs sur GPU pour chaque block
	for(int i = 0; i < NB*NTPB; i++){
		testCUDA(hipMemcpy(aGPU + i*(n-1), &(a[1]), (n-1)*sizeof(float), hipMemcpyHostToDevice));
		testCUDA(hipMemcpy(bGPU + i*n, b, n*sizeof(float), hipMemcpyHostToDevice));
		testCUDA(hipMemcpy(cGPU + i*(n-1), &(c[1]), (n-1)*sizeof(float), hipMemcpyHostToDevice));
		testCUDA(hipMemcpy(yGPU + i*n, y, n*sizeof(float), hipMemcpyHostToDevice));
	}

	Thomas<<<NB, NTPB>>>(aGPU, bGPU, cGPU, yGPU, zGPU, n);

	for(int i = 0; i < NB*NTPB; i++){
		testCUDA(hipMemcpy(z, zGPU + i*n, n*sizeof(float), hipMemcpyDeviceToHost));
		printVect(z, n);
	}

	// Libération des vecteurs GPU
	testCUDA(hipFree(aGPU));
	testCUDA(hipFree(bGPU));
	testCUDA(hipFree(cGPU));
	testCUDA(hipFree(yGPU));
	testCUDA(hipFree(zGPU));
}

void PCR_wrap(float* a, float* b, float* c, float* y, int* z, int n){
	// Déclaration des variables utilisées
	float *aGPU, *bGPU, *cGPU, *yGPU;
	int *zGPU;

	// Allocation des vecteurs dans la mémoire GPU
	testCUDA(hipMalloc(&aGPU, NB*n*sizeof(float)));  	//sa
	testCUDA(hipMalloc(&cGPU, NB*n*sizeof(float)));  	//sc
	testCUDA(hipMalloc(&bGPU, NB*n*sizeof(float)));		//sd
	testCUDA(hipMalloc(&yGPU, NB*n*sizeof(float)));		//sy (contains the solution after calling PCR_d)
	testCUDA(hipMalloc(&zGPU, NB*n*sizeof(int)));			//sl

	// Copie des données dans les vecteurs sur GPU pour chaque block
	for(int i = 0; i < NB; i++){
		testCUDA(hipMemcpy(aGPU + i*n, a, n*sizeof(float), hipMemcpyHostToDevice));
		testCUDA(hipMemcpy(bGPU + i*n, b, n*sizeof(float), hipMemcpyHostToDevice));
		testCUDA(hipMemcpy(cGPU + i*n, c, n*sizeof(float), hipMemcpyHostToDevice));
		testCUDA(hipMemcpy(yGPU + i*n, y, n*sizeof(float), hipMemcpyHostToDevice));
	}

	PCR<<<NB, NTPB>>>(aGPU, bGPU, cGPU, yGPU, zGPU, n);

	for(int i = 0; i < NB; i++){
		testCUDA(hipMemcpy(y, yGPU + i*n, n*sizeof(float), hipMemcpyDeviceToHost));	// !!! SOLUTION IN yGPU !!!
		printVect(y, n);
	}

	// Libération des vecteurs GPU
	testCUDA(hipFree(aGPU));
	testCUDA(hipFree(bGPU));
	testCUDA(hipFree(cGPU));
	testCUDA(hipFree(yGPU));
	testCUDA(hipFree(zGPU));
}

int main(void){
	int n = N;

	float *a, *b, *c, *y, *z;
	int *pcr_z;

	// Allocation des vecteurs pour initialisation du système
	a = (float *) malloc(n* sizeof(float) );
	b = (float *) malloc(n * sizeof(float) );
	c = (float *) malloc(n * sizeof(float) );
	y = (float *) malloc(n * sizeof(float) );
	z = (float *) malloc(n * sizeof(float) );
	pcr_z = (int *) malloc(n * sizeof(int) );

	iniTest(a, b, c, y, n);

	a[0] = 0.f;
	c[0] = 0.f;

	printf("Vecteur a :\n");
	printVect(a, n);
	printf("Vecteur b :\n");
	printVect(b, n);
	printf("Vecteur c :\n");
	printVect(c, n);
	printf("Vecteur y :\n");
	printVect(y, n);

	//Test Thomas
	Thomas_wrap(a, b, c, y, z, n);

	//Test PCR
	//PCR_wrap(a, b, c, y, pcr_z, n);

	// Libération des vecteurs sur RAM
	free(a);
	free(b);
	free(c);
	free(y);
	free(z);
	free(pcr_z);

	return 0;
}